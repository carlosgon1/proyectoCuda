#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utilityarray.h"
#include "operations.h" 
#define R 5636
#define C 1000

typedef struct
{
	float minD;				//GUARDA EL MINIMO DEL VECTOR.
	int md;					//GUARDA EL INDICE DONDE ENCONTRO EL MINIMO.
}Particules;

void calAllForcesGravitational_GPU(float *h_Forces,float *h_dist,float *h_mass,float *h_spacePoints,int M,int N);
void checkCUDAError(const char *msg);
__global__ void kernel(float *F,float *points,float *dist,float *mass,int M,int numVar);
Particules min(float *vector,int SIZE);


float x1[R][C]={ 	
	
int main()
{  
    int i,k,j;
    int J=R;
    size_t sizeM = J * sizeof(float);     
    size_t sizeN= C* sizeof(float); 
   	int SIZE=J;
	int INTERACTIONS_EXIST = 1;
	int iters=0;
	float radius = 0.5;
	float modifier = 0.9;
    float d, distp, t;
	int op1, op2, a, b;
 	
 	// DECLARACION E INICIALIZACION DE VARIABLES DEL TEMPORIZADOR EN EL GPU
  	float time; 
  	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //////////////////////////////////////

    float *x;
    x=(float*)malloc(sizeM*sizeN);

    float *F;
    F = (float*)malloc(sizeM*sizeM);  

    float *dist;
    dist= (float*)malloc(sizeM*sizeM);  

    float *m;
    m= (float*)malloc(sizeM); 

    int *index;
    initializateArray(m,J,100);
    //COPY ARRAY GLOBAL TO ARRAY LOCAL
    for(i=0;i<R;i++)
    {
        for(j=0;j<C;j++)
        {
            x[i*C+j]=x1[i][j];
        }
    }
  
	while(INTERACTIONS_EXIST ==1)
	{
		 
		INTERACTIONS_EXIST = 0;
		iters = iters + 1;
		initializateArray(F, SIZE*SIZE, 0);
		initializateArray(dist, SIZE*SIZE, 0);

		/***Calculate all interacting forces in the system***/
		calAllForcesGravitational_GPU(F,dist,m,x,SIZE,C);
		/*hipEventRecord(start, 0);
		calAllForcesGravitational_CPU(F,dist,m,x,SIZE,C);
		hipEventRecord(stop, 0);    
    	hipEventSynchronize(stop); 
    	hipEventElapsedTime( &time, start, stop );
    	printf("TIEMPO DE EJECUCION EN CPU: %f ms\n",time);
    	getchar();
		/******************Reorder particule Data*******************/
		//printMatrix(F,SIZE,SIZE,"FUERZAS GRAVITACIONALES CALCULADAS");
		index=SortSumAllColumns(F,SIZE);
		//printArray(index,SIZE,"INDICE DE PARTICULAS ORDENADAS");
		/******************Unificate Particules*********************/
		int newSize=SIZE;
		for(i=0;i<SIZE;i++)
		{
			int idx=index[i];
			Particules p=min(dist+(idx*SIZE),SIZE);
			op1=idx;
			op2=p.md;
			if(m[op1]!=-1 && m[op2]!=-1 && op1!=op2 && p.minD<radius)
			{
				INTERACTIONS_EXIST=1;
				if(m[op1] >= m[op2])
				{
					a=op1;
					b=op2;
				}
				else
				{
					a=op2;
					b=op1;
				}
				m[a]=m[a]+m[b];
				/******************move particulas according to masses************/
				float dSum;
				for(k=0;k<C;k++)
				{
					dSum= dSum + pow((x[a*C+k]-x[b*C + k]),2);
				}
				d=sqrt(dSum);
				distp=(m[b] / (m[a]+m[b]))*d;
				t=distp/d;
				for (j = 0; j<C; j++)
				{
					x[a*C+j] = x[a*C + j] + t*(x[b*C + j] - x[a*C + j]);
				}

				m[b]=-1;
				x[b*C] = -2;
				newSize--;
			}
			 
		}
		free(index);

		/******************CLEAN UP ELEMENT FROM MASS AND SPACE POINTS*************/
		clearMass(m,SIZE);
		clearSpacePoints(x,SIZE,C);

		//printArrayfloat(m,newSize,"VALORES DE LAS MASAS");
		//printMatrix(x,newSize,C,"PUNTOS EN EL ESPACIOS");
		radius=radius*modifier;
		SIZE=newSize;
	}
	//printMatrix(x,SIZE,C,"RESULTADO FINAL");
	hipEventDestroy( start ); // GC DEL TEMPORIZADOR
    hipEventDestroy( stop );  // GC DEL TEMPORIZADOR
	free(m);
	free(x);
	free(F);
	free(dist);
	 
	return 0;
}

 

Particules min(float *vector,int size)
 {
 	Particules p;
 	p.minD=0;
 	p.md=0;
 	int i;
 	for(i=0;i<size;i++)
	{
		if(p.minD==0)
		{
			p.minD=vector[i];
			p.md=i;
		}
		else if(vector[i]<p.minD)
		{
			p.minD=vector[i];	
			p.md=i;
		}
	}
	return p;
 }




void calAllForcesGravitational_GPU(float *h_Forces,float *h_dist,float *h_mass,float *h_spacePoints,int M,int N)
{
	size_t sizeN = N * sizeof(float); 			 
    size_t sizeM = M * sizeof(float); 
    // GPU'S TIMER VARIABLES
  	float time; 
  	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //////////////////////////////////////

    /*CONFIGURATION FOR KERNEL FOR TO WORK WHIT MATRIX 5632 x 5632 */
    dim3 grid(256,256);
    dim3 block(22,22);

    /*CREATE MATRIX WHERE SAVE ALL VALUES OF ALL FORCES CALCULATED*/
	float *d_Forces;
    hipMalloc((void **)&d_Forces,sizeM*sizeM);
    hipMemset(d_Forces,0,sizeM*sizeM); 	

    /*CREATE MATRIX WHERE SAVE ALL VALUES OF THE DISTANCES*/
	float *d_dist;
    hipMalloc((void **)&d_dist,sizeM*sizeM);
    hipMemset(d_dist,0,sizeM*sizeM); 					

	/*COPY VALUES OF SPACEPOINTS OF HOST TO DEVICE*/
    float *d_spacePoints;
    hipMalloc((void **)&d_spacePoints,sizeM*sizeN);
    hipMemset(d_spacePoints,0,sizeM*sizeN); 
    hipMemcpy(d_spacePoints,h_spacePoints,sizeM*sizeN,hipMemcpyHostToDevice);

    /*COPY VALUES OF MASS OF HOST TO DEVICE*/
    float *d_mass;
    hipMalloc((void **)&d_mass,sizeM);
    hipMemcpy(d_mass,h_mass,sizeM,hipMemcpyHostToDevice);
    /*********************APPLY TIMER***********************************/
    hipEventRecord(start, 0); 
    kernel<<< grid,block>>>(d_Forces,d_spacePoints,d_dist,d_mass,M,N);
	hipEventRecord(stop, 0);    
    hipEventSynchronize(stop); 
    hipEventElapsedTime( &time, start, stop );
	hipDeviceSynchronize();
	printf("TIEMPO DE EJECUCION EN GPU: %f ms\n",time);
    getchar();

    /*RECOVERY FORCES AND DISTANCE TO HOST*/
    hipMemcpy(h_Forces,d_Forces,sizeM*sizeM,hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy");
    hipMemcpy(h_dist,d_dist,sizeM*sizeM,hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy");

    hipFree(d_Forces);
    hipFree(d_mass);
    hipFree(d_dist);
    hipFree(d_spacePoints);
}


__global__ void kernel(float *F,float *points,float *dist,float *mass,int M,int numVar)
{
	int i;
	float rest,dSum;
	int tidx = (blockDim.x * blockIdx.x) + threadIdx.x;
	int tidy = (blockDim.y * blockIdx.y) + threadIdx.y;
    if(tidx <M && tidy<M)
    {
    	if( tidx != tidy )
     	{
     		dSum=0;
     		for( i=0; i < numVar; i++ )
     		{
    			rest = (points[(tidx * numVar) + i]-points[(tidy * numVar) + i]);
    			dSum= dSum + (rest*rest);//al cuadrado
     		}
     		dist[tidx*M+tidy] = sqrt(dSum);
     		F[tidx*M+tidy] = (G * mass[tidx] * mass[tidy]) / dSum;
     	} 
     	else
     	{
     		dist[tidx*M+tidy]=3;
     	}
    }
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}